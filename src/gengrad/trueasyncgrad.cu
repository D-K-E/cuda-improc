#include "hip/hip_runtime.h"
// simple
#include <cudaimproc/cudacheck.h>
#include <cudaimproc/execonfig.h>
#include <cudaimproc/imgio.h>
//
#include <hip/hip_runtime.h>
#include <iostream>
#include <optional>

namespace cudaimproc {
__global__ void
gen_gradient_async(unsigned char *pixels, const int imwidth,
                   const int imheight,
                   const std::size_t stream_offset) {
  //
  int col = threadIdx.x; // local thread index
  col +=
      blockIdx.x * blockDim.x; // thread block id * number
                               // of threads_per_block
  if (col >= imwidth) {
    return;
  }

  if (rgb_offset == 2) {
  }
  int bytes_per_pixel = 3;
  int per_scanline = imwidth * bytes_per_pixel;

  float g = static_cast<float>(col) /
            static_cast<float>(imwidth - 1);
  unsigned char green =
      static_cast<unsigned char>(g * 255.99);

  float b = 0.25f;
  unsigned char blue = static_cast<unsigned char>(b * 255.99);

  for (int row = 0; row < imheight; ++row) {
    //
    float r = static_cast<float>(row) /
              static_cast<float>(imheight - 1);
    unsigned char red =
        static_cast<unsigned char>(r * 255.99);
    // TODO write offset code
  }
}
}; // namespace cudaimproc

int main(void) { // yep this is (void) type of day
                 //

  // image config
  const float aspect_ratio = 16.0f / 9.0f;
  const std::size_t imwidth = 640;
  const std::size_t imheight = static_cast<int>(
      static_cast<float>(imwidth) / aspect_ratio);
  const std::size_t bytes_per_line = imwidth * 3;
  const std::size_t imsize = bytes_per_line * imheight;
  const std::size_t imsizeInBytes =
      imsize * sizeof(unsigned char);

  // kernel config

  const std::size_t threads_per_block = 64;
  const std::size_t nb_streams =
      3; // 1 for each rgb component
  cudaimproc::ExecutionConfig1D config(
      imheight, threads_per_block, nb_streams);
  const std::size_t streamSize =
      imsize / config.nb_streams();
  hipStream_t streams[nb_streams];

  // create cuda stream
  for (int i = 0; i < nb_streams; ++i) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  unsigned char *pixels_device{nullptr};
  //
  // cuda malloc
  CUDA_CHECK(
      hipMalloc((void **)(&pixels_device), imsizeInBytes));

  //

  for (int i = 0; i < nb_streams; ++i) {
    std::size_t sblock_nb =
        streamSize / config.nb_threads();
    cudaimproc::gen_gradient_async<<<
        sblock_nb, config.nb_threads(), 0, streams[i]>>>(
        pixels_device, imwidth, imheight, i);
    CUDA_CHECK(hipGetLastError());
  }
  CUDA_CHECK(hipDeviceSynchronize());
  //
  unsigned char *pixels_host = new unsigned char[imsize];
  CUDA_CHECK(hipMemcpy(pixels_host, pixels_device,
                        imsizeInBytes,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(pixels_device));

  //
  cudaimproc::render(std::make_optional(pixels_host),
                     imheight, imwidth, 3, "asyncimg");
  // destroy resources
  for (int i = 0; i < nb_streams; ++i) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
  delete[] pixels_host;
  return 0;
}
